#include "hip/hip_runtime.h"
#include <assert.h>

#include "cudaMVNormal.hu"

/*
 * Computes \sum_{i}^{N} x_i y_i for x, y \in \mathbb{R}^{N}.
 */
__device__ double devVecDot(const size_t N, const double* x, const double* y) {
	assert(N > 0);
	assert(x != NULL);
	assert(y != NULL);
	// x == y allowed

	double sum = 0;
	for(size_t i = 0; i < N; ++i) {
		sum += x[i] * y[i];
	}
	return sum;
}

/*
 * Computes z_{i} \gets x_{i} - y_{i} for x, y \in \mathbb{R}^N.
 */
__device__ void devVecMinus(const size_t N, double* z, const double* x, const double* y) {
	assert(N > 0);
	assert(x != NULL);
	assert(y != NULL);
	// x == y allowed

	for(size_t i = 0; i < N; ++i) {
		z[i] = x[i] - y[i];
	}
}

/*
 * Solves the lower triangular system L^T x = b for x, b \in \mathbb{R}^{N}, 
 * L \in \mathbb{R}^{N \times N} and L_{i, j} = 0 for j > i.
 */
__device__ void devSolveLowerTri(const size_t N, const double* L, double* x, const double* b) {
	assert(N > 0);
	assert(L != NULL);
	assert(x != NULL);
	assert(b != NULL);
	// x == b allowed

	for(size_t i = 0; i < N; ++i) {
		double sum = 0.0;
		if(i > 0) {
			for(size_t j = 0; j <= i - 1; ++j) {
				sum += L[i * N + j] * x[j];
			}
		}

		x[i] = (b[i] - sum) / L[i * N + i];
	}
}

/*
 * Solves the upper triangular system L^T x = b for x, b \in \mathbb{R}^{N}, 
 * L \in \mathbb{R}^{N \times N} and L_{i, j} = 0 for j > i.
 */
__device__ void devSolveLowerTriT(const size_t N, const double* L, double* x, const double* b) {
	assert(N > 0);
	assert(L != NULL);
	assert(x != NULL);
	assert(b != NULL);
	// x == b allowed

	// treat L as an upper triangular matrix U
	for(size_t i = 0; i < N; i++) {
		size_t ip = N - 1 - i;
		double sum = 0;
		for(size_t j = ip + 1; j < N; ++j) {
			sum += L[j * N + ip] * x[j];
		}

		x[ip] = (b[ip] - sum) / L[ip * N + ip];
	}
}


/*
 *
 */
__device__ double devLogMVNormNormalizer(
	const size_t pointDim,
	const double* sigmaL
) {
	double det = 1.0;
	for(size_t i = 0; i < pointDim; ++i) {
		det *= sigmaL[i * pointDim + i];
	}
	det *= det;

	return -0.5 * log( 2.0 * M_PI ) * pointDim - 0.5 * log(det);
}

/*
 * Computes log( p(x | mu, Sigma ) ) for multivariate normal distribution with 
 * parameters mu (mean), and Sigma (covariance).
 */
__device__ double devLogMVNormDist(
	const size_t pointDim,
	const double* x, const double* mu, const double* sigmaL,
	double* u, double* v
) {
	devVecMinus(pointDim, v, x, mu); // v <- x - mu
	devSolveLowerTri(pointDim, sigmaL, u, v); // u <- u s.t. L u = (x - mu)
	devSolveLowerTriT(pointDim, sigmaL, u, u); // u <- v s.t. L^T v = u
	return devLogMVNormNormalizer(pointDim, sigmaL) - 0.5 * devVecDot(pointDim, u, v);
}

__global__ void kernLogMVNormDist(
	const size_t numPoints, const size_t pointDim, 
	const double* X, double* mu, double* sigmaL,
	double* logProb
) {
	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;
	if(i >= numPoints) {
		return;
	}

	double u[1024];
	double v[1024];

	logProb[i] = devLogMVNormDist(
		pointDim, 
		& X[i * pointDim], mu, sigmaL,
		u, v
	);
}

