#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdlib.h>

#include "cudaGmm.hu"

__global__ void kernGmmLogLikelihood(
	const size_t numPoints, const size_t numComponents,
	const double* logPi, const double* logP,
	double* logL
) {
	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;
	if(i >= numPoints) {
		return;
	}

	double maxArg = -INFINITY;
	for(size_t k = 0; k < numComponents; ++k) {
		const double logProbK = logPi[k] + logP[k * numPoints + i];
		if(logProbK > maxArg) {
			maxArg = logProbK;
		}
	}

	double sum = 0.0;
	for (size_t k = 0; k < numComponents; ++k) {
		const double logProbK = logPi[k] + logP[k * numPoints + i];
		sum = exp(logProbK - maxArg);
	}

	logL[i] = maxArg + log(sum);
}
