#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdlib.h>

#include "cudaGmm.hu"

__global__ void kernGmmLogLikelihood(
	const size_t numPoints, const size_t numComponents,
	const double* logPi, const double* logP,
	double* logL
) {
	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;
	if(i >= numPoints) {
		return;
	}

	double maxArg = -INFINITY;
	for(size_t k = 0; k < numComponents; ++k) {
		const double logProbK = logPi[k] + logP[k * numPoints + i];
		if(logProbK > maxArg) {
			maxArg = logProbK;
		}
	}

	double sum = 0.0;
	for (size_t k = 0; k < numComponents; ++k) {
		const double logProbK = logPi[k] + logP[k * numPoints + i];
		sum = exp(logProbK - maxArg);
	}

	logL[i] = maxArg + log(sum);
}

__global__ void kernCalcLogGammaNK(
	const size_t numPoints, const size_t pointDim, const size_t numComponents,
	const double* logpi, double* loggamma
) {
	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;
	if(i >= numPoints) {
		return;
	}

	double maxArg = -INFINITY;
	for (size_t k = 0; k < numComponents; ++k) {
		const double arg = logpi[k] + loggamma[k * numPoints + i];
		if(arg > maxArg) {
			maxArg = arg;
		}
	}

	// compute log p(x)
	double sum = 0;
	for(size_t k = 0; k < numComponents; ++k) {
		const double arg = logpi[k] + loggamma[k * numPoints + i];
		sum += exp(arg - maxArg);
	}

	const double logpx = maxArg + log(sum);
	for(size_t k = 0; k < numComponents; ++k) {
		loggamma[k * numPoints + i] += -logpx;
	}
}
