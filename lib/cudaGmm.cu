#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include "cudaCommon.hu"
#include "cudaFolds.hu"
#include "cudaGmm.hu"

__global__ void kernGmmLogLikelihood(
	const size_t numPoints, const size_t numComponents,
	const double* logPi, const double* logP,
	double* logL
) {
	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;
	if(i >= numPoints) {
		return;
	}

	double maxArg = -INFINITY;
	for(size_t k = 0; k < numComponents; ++k) {
		const double logProbK = logPi[k] + logP[k * numPoints + i];
		if(logProbK > maxArg) {
			maxArg = logProbK;
		}
	}

	double sum = 0.0;
	for (size_t k = 0; k < numComponents; ++k) {
		const double logProbK = logPi[k] + logP[k * numPoints + i];
		sum = exp(logProbK - maxArg);
	}

	logL[i] = maxArg + log(sum);
}

__host__ double cudaGmmLogLikelihood(
	hipDeviceProp_t* deviceProp,
	const size_t numPoints, const size_t numComponents,
	const size_t M,
	const double* logpi, const double* logP,
	const double* device_logpi, const double* device_logP
) {
	dim3 grid, block;
	calcDim(M, deviceProp, &block, &grid);

	double logL = 0;
	double* device_logL = mallocOnGpu(M);

	kernGmmLogLikelihood<<<grid, block>>>(
		M, numComponents,
		device_logpi, device_logP, device_logL
	);

	cudaArraySum(
		deviceProp, 
		M, 1, 
		device_logL, 
		&logL
	);

	hipFree(device_logL);

	if(M != numPoints) {
		for(size_t i = M; i < numPoints; ++i) {
			double maxArg = -INFINITY;
			for(size_t k = 0; k < numComponents; ++k) {
				const double logProbK = logpi[k] + logP[k * numPoints + i];
				if(logProbK > maxArg) {
					maxArg = logProbK;
				}
			}

			double sum = 0.0;
			for (size_t k = 0; k < numComponents; ++k) {
				const double logProbK = logpi[k] + logP[k * numPoints + i];
				sum = exp(logProbK - maxArg);
			}

			logL += maxArg + log(sum);
		}
	}

	return logL;
}

__global__ void kernCalcLogGammaNK(
	const size_t numPoints, const size_t pointDim, const size_t numComponents,
	const double* logpi, double* loggamma
) {
	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;
	if(i >= numPoints) {
		return;
	}

	double maxArg = -INFINITY;
	for (size_t k = 0; k < numComponents; ++k) {
		const double arg = logpi[k] + loggamma[k * numPoints + i];
		if(arg > maxArg) {
			maxArg = arg;
		}
	}

	// compute log p(x)
	double sum = 0;
	for(size_t k = 0; k < numComponents; ++k) {
		const double arg = logpi[k] + loggamma[k * numPoints + i];
		sum += exp(arg - maxArg);
	}

	const double logpx = maxArg + log(sum);
	for(size_t k = 0; k < numComponents; ++k) {
		loggamma[k * numPoints + i] += -logpx;
	}
}

__global__ void kernCalcMu(
	const size_t numPoints, const size_t pointDim,
	const double* X, const double* loggamma, const double logGammaK,
	double* dest
) {
	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;
	if(i >= numPoints) {
		return;
	}

	const double a = exp(loggamma[i]) / exp(logGammaK);
	const double* x = & X[i * pointDim];
	double* y = & dest[i * pointDim]; 

	for(size_t i = 0; i < pointDim; ++i) {
		y[i] = a * x[i];
	}
}

__global__ void kernCalcSigma(
	const size_t numPoints, const size_t pointDim,
	const double* X, const double* mu, const double* loggamma, const double logGammaK,
	double* dest
) {
	assert(pointDim < 1024);
	
	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;
	if(i >= numPoints) {
		return;
	}

	// gamma_{n,k} / Gamma_{k} (x - mu) (x - mu)^T

	const double a = exp(loggamma[i]) / exp(logGammaK);
	const double* x = & X[i * pointDim];
	double* y = & dest[i * pointDim * pointDim]; 

	double u[1024];
	for(size_t i = 0; i < pointDim; ++i) {
		u[i] = x[i] - mu[i];
	}

	for(size_t i = 0; i < pointDim; ++i) {
		double* yp = &y[i * pointDim];
		for(size_t j = 0; j < pointDim; ++j) {
			yp[j] = a * u[i] * u[j];
		}
	}
}

__host__ void cudaUpdateMu(
	hipDeviceProp_t* deviceProp,
	const size_t numPoints, const size_t pointDim,
	const size_t M,
	const double* X, const double* loggamma, const double logGammaK,
	const double* device_X, const double* device_loggamma,
	double* mu
) {
	dim3 grid, block;
	calcDim(M, deviceProp, &block, &grid);

	double* device_mu_working = mallocOnGpu(M * pointDim);

	kernCalcMu<<<grid, block>>>(
		M, pointDim, 
		device_X, device_loggamma, logGammaK, 
		device_mu_working
	);

	cudaArraySum(
		deviceProp,
		M, pointDim, 
		device_mu_working,
		mu
	);

	hipFree(device_mu_working);

	if(M != numPoints) {
		double cpuMuSum[pointDim];
		memset(cpuMuSum, 0, pointDim * sizeof(double));
		for(size_t i = M; i < numPoints; ++i) {
			double a = exp(loggamma[i]) / exp(logGammaK);
			for(size_t j = 0; j < pointDim; ++j) {
				cpuMuSum[j] += a * X[i * pointDim + j];
			}
		}

		for(size_t i = 0; i < pointDim; ++i) {
			mu[i] += cpuMuSum[i];
		}
	}
}

__host__ void cudaUpdateSigma(
	hipDeviceProp_t* deviceProp,
	const size_t numPoints, const size_t pointDim,
	const size_t M,
	const double* X, const double* loggamma, const double logGammaK,
	const double* device_X, const double* device_loggamma,
	double* mu, 
	double* sigma
) {
	dim3 grid, block;
	calcDim(M, deviceProp, &block, &grid);

	double* device_mu = sendToGpu(pointDim, mu);
	double* device_sigma_working = mallocOnGpu(M * pointDim * pointDim);

	kernCalcSigma<<<grid, block>>>(
		M, pointDim, 
		device_X, device_mu, device_loggamma, logGammaK, 
		device_sigma_working
	);

	cudaArraySum(
		deviceProp,
		M, pointDim * pointDim, 
		device_sigma_working,
		sigma
	);

	hipFree(device_sigma_working);

	if(M != numPoints) {
		double cpuSigmaSum[pointDim * pointDim];
		memset(cpuSigmaSum, 0, pointDim * pointDim * sizeof(double));

		for(size_t i = M; i < numPoints; ++i) {
			double a = exp(loggamma[i]) / exp(logGammaK);
	
			double xm[pointDim];
			for(size_t j = 0; j < pointDim; ++j) {
				xm[j] = X[i * pointDim + j] - mu[j]; 
			}

			for(size_t j = 0; j < pointDim; ++j) {
				for(size_t k = 0; k < pointDim ; ++k) {
					cpuSigmaSum[j * pointDim + k] += a * xm[j] * xm[k];
				}
			}
		}

		for(size_t i = 0; i < pointDim * pointDim; ++i) {
			sigma[i] += cpuSigmaSum[i];
		}
	}
}
