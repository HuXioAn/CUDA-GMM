#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include "cudaCommon.hu"

__host__ void assertPowerOfTwo(size_t N) {
	int bit = 0;
	while(N > 0 && bit <= 1) {
		bit += N & 1;
		N >>= 1;
	}

	assert(bit <= 1);
}

__host__ size_t largestPowTwoLessThanEq(size_t N) {
	// Assigns the largest value (M = 2^n) < N to N and returns the residual.
	if(N == 0) {
		return 0;
	} // PC: N > 0

	size_t M = 1;
	while(M < N) {
		M *= 2;
	} // PC: M >= N

	if(M == N) {
		return M;
	} // PC: M > N
	
	return M / 2;
}

__host__ void calcDim(int N, hipDeviceProp_t* devProp, dim3* block, dim3* grid) {
	assert(devProp != NULL);
	assert(block != NULL);
	assert(grid != NULL);

	// make a 2D grid of 1D blocks
	const int numThreadRows = 1;
	const int numThreadCols = devProp->maxThreadsPerBlock;
	block->x = min(numThreadCols, N);
	block->y = numThreadRows;

	const int numThreadsPerBlock = numThreadRows * numThreadCols;
	const int residualThreads = N % numThreadsPerBlock;
	int numBlocksPerGrid = (N - residualThreads) / numThreadsPerBlock;
	if(residualThreads > 0) {
		++numBlocksPerGrid;
	}

	const int numBlockCols = min( numBlocksPerGrid, devProp->maxGridSize[0] );
	const int residualBlocks = numBlocksPerGrid % numBlockCols;
	int numBlockRows = (numBlocksPerGrid - residualBlocks) / numBlockCols;
	if(residualBlocks > 0) {
		++numBlockRows;
	}

	grid->x = numBlockCols;
	grid->y = numBlockRows;

	assert(block->x * block->y * block->z > 0);
	assert(block->x * block->y * block->z <= devProp->maxThreadsPerBlock);

	assert(grid->x <= devProp->maxGridSize[0]);
	assert(grid->y <= devProp->maxGridSize[1]);
	assert(grid->z <= devProp->maxGridSize[2]);
}

__host__ void dimToConsole(dim3* block, dim3* grid) {
	assert(block != NULL);
	assert(grid != NULL);

	printf("block: (%d, %d, %d)\n", block->x, block->y, block->z);
	printf("grid: (%d, %d, %d)\n", grid->x, grid->y, grid->z);
}

__host__ double* mallocOnGpu(const size_t N) {
	double* device_A;
	double ABytes = N * sizeof(double);
	check(hipMalloc(&device_A, ABytes));
	return device_A;
}

__host__ double* sendToGpu(const size_t N, const double* A) {
	double* device_A;
	const size_t ABytes = N * sizeof(double);
	check(hipMalloc(&device_A, ABytes));
	check(hipMemcpy(device_A, A, ABytes, hipMemcpyHostToDevice));
	return device_A;
}

__device__ void devVecAdd(size_t pointDim, double* dest, double* src) {
	for(size_t i = 0; i < pointDim; ++i) {
		dest[i] += src[i];
	}
}

__global__ void kernElementWiseSum(const size_t numPoints, const size_t pointDim, double* dest, double* src) {
	// Assumes a 2D grid of 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;

	if(i < numPoints) {
		devVecAdd(pointDim, &dest[i * pointDim], &src[i * pointDim]);
	}
}

__global__ void kernBlockWiseSum(const size_t numPoints, const size_t pointDim, double* dest) {
	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;

	if(i < numPoints) {
		// Do all the calculations in block shared memory instead of global memory.
		for(int s = blockDim.x / 2; threadIdx.x < s; s /= 2) {
			devVecAdd(pointDim, &dest[i * pointDim], &dest[(i + s) * pointDim]);
			__syncthreads();
		}
	}
}

__host__ void cudaArraySum(hipDeviceProp_t* deviceProp, size_t numPoints, const size_t pointDim, double* device_A) {
	assert(deviceProp != NULL);
	assert(numPoints > 0);
	assertPowerOfTwo(numPoints);
	assert(pointDim > 0);
	assert(device_A != NULL);

	// Parallel sum by continually folding the array in half and adding the right 
	// half to the left half until the fold size is 1024 (single block), then let
	// GPU reduce the remaining block to a single value and copy it over. O(log n).
	if(numPoints > 1024) {
		dim3 block, grid;
		for(numPoints /= 2; numPoints >= 1024; numPoints /= 2) {
			calcDim(numPoints, deviceProp, &block, &grid);
			kernElementWiseSum<<<grid, block>>>(
				numPoints, pointDim, device_A, device_A + numPoints * pointDim
			);
		}
		numPoints *= 2;
	}
 
	kernBlockWiseSum<<<1, numPoints>>>(
		numPoints, pointDim, device_A
	);
}

__host__ void cudaArraySum(hipDeviceProp_t* deviceProp, size_t numPoints, const size_t pointDim, double* device_A, double* host_sum) {
	assert(host_sum != NULL);
	cudaArraySum(deviceProp, numPoints, pointDim, device_A);
	check(hipMemcpy(host_sum, device_A, pointDim * sizeof(double), hipMemcpyDeviceToHost));
}

__global__ void kernElementWiseMax(int N, double* dest, double* src) {
	// Assumes a 2D grid of 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;
	if(i < N) {
		if(dest[i] < src[i]) {
			dest[i] = src[i];
		}
	}
}

__global__ void kernBlockWiseMax(const size_t N, double* dest) {
	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;

	// Load into block shared memory
	__shared__ double blockMax[1024];

	if(threadIdx.x >= N) {
		blockMax[threadIdx.x] = -INFINITY;
	} else {
		blockMax[threadIdx.x] = dest[i];
	}

	__syncthreads();	

	// Do all the calculations in block shared memory instead of global memory.
	for(int s = blockDim.x / 2; threadIdx.x < s; s /= 2) {
		if(blockMax[threadIdx.x] < blockMax[threadIdx.x + s]) {
			blockMax[threadIdx.x] = blockMax[threadIdx.x + s];
		}
		__syncthreads();
	}

	if(threadIdx.x == 0) {
		// Just do one global write instead of 2048.
		dest[i] = blockMax[0];
	}
}

__host__ double cudaArrayMax(hipDeviceProp_t* deviceProp, const size_t N, double* device_A) {
	// Parallel max by continually folding the array in half and maxing the right 
	// half to the left half until the fold size is 1024 (single block), then let
	// GPU reduce the remaining block to a single value and copy it over. O(log n).
	if(N > 1024) {
		assertPowerOfTwo(N);
		dim3 block, grid;
		for(size_t n = N/2; n >= 1024; n /= 2) {
			calcDim(n, deviceProp, &block, &grid);
			kernElementWiseMax<<<grid, block>>>(n, device_A, device_A + n);
			check((void)0);
		}
	}
 
	kernBlockWiseMax<<<1, 1024>>>(N, device_A);
	check((void)0);

	double maxValue = 0;
	check(hipMemcpy(&maxValue, device_A, sizeof(double), hipMemcpyDeviceToHost));
	check(hipDeviceSynchronize());
	return maxValue;
}
