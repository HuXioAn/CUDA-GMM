#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include "cudaCommon.hu"

__host__ void calcDim(int N, hipDeviceProp_t* devProp, dim3* block, dim3* grid) {
	assert(devProp != NULL);
	assert(block != NULL);
	assert(grid != NULL);

	// make a 2D grid of 1D blocks
	const int numThreadRows = 1;
	const int numThreadCols = devProp->maxThreadsPerBlock;
	block->x = min(numThreadCols, N);
	block->y = numThreadRows;

	const int numThreadsPerBlock = numThreadRows * numThreadCols;
	const int residualThreads = N % numThreadsPerBlock;
	int numBlocksPerGrid = (N - residualThreads) / numThreadsPerBlock;
	if(residualThreads > 0) {
		++numBlocksPerGrid;
	}

	const int numBlockCols = min( numBlocksPerGrid, devProp->maxGridSize[0] );
	const int residualBlocks = numBlocksPerGrid % numBlockCols;
	int numBlockRows = (numBlocksPerGrid - residualBlocks) / numBlockCols;
	if(residualBlocks > 0) {
		++numBlockRows;
	}

	grid->x = numBlockCols;
	grid->y = numBlockRows;

	assert(block->x * block->y * block->z > 0);
	assert(block->x * block->y * block->z <= devProp->maxThreadsPerBlock);

	assert(grid->x <= devProp->maxGridSize[0]);
	assert(grid->y <= devProp->maxGridSize[1]);
	assert(grid->z <= devProp->maxGridSize[2]);
}

__host__ void dimToConsole(dim3* block, dim3* grid) {
	assert(block != NULL);
	assert(grid != NULL);

	printf("block: (%d, %d, %d)\n", block->x, block->y, block->z);
	printf("grid: (%d, %d, %d)\n", grid->x, grid->y, grid->z);
}

__host__ void assertPowerOfTwo(size_t N) {
	int bit = 0;
	while(N > 0 && bit <= 1) {
		bit += N & 1;
		N >>= 1;
	}
	assert(bit <= 1);
}

__global__ void kernArraySum(int N, double* dest, double* src) {
	// Assumes a 2D grid of 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;
	dest[i] += src[i];
}

__global__ void kernReduceBlocks(double* dest) {
	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;

	// Load into block shared memory
	__shared__ double localSum[1024];
	localSum[threadIdx.x] = dest[i];
	__syncthreads();	

	// Do all the calculations in block shared memory instead of global memory.
	for(int s = blockDim.x / 2; threadIdx.x < s; s /= 2) {
		localSum[threadIdx.x] += localSum[threadIdx.x + s];
		__syncthreads();
	}

	if(threadIdx.x == 0) {
		// Just do one global write instead of 2048.
		dest[i] = localSum[0];
	}
}

__host__ double cudaReduceSum(hipDeviceProp_t* deviceProp, const size_t N, double* device_A) {
	// Parallel sum by continually folding the array in half and adding the right 
	// half to the left half until the fold size is 1024 (single block), then let
	// GPU reduce the remaining block to a single value and copy it over. O(log n).
	if(N >= 1024) {
		dim3 block, grid;
		for(size_t n = N/2; n >= 1024; n /= 2) {
			calcDim(n, deviceProp, &block, &grid);
			kernArraySum<<<grid, block>>>(n, device_A, device_A + n);
		}
		kernReduceBlocks<<<1, 1024>>>(device_A);
	} else {
		kernReduceBlocks<<<1, N>>>(device_A);
	}

	double sum = 0;
	check(hipMemcpy(&sum, device_A, sizeof(double), hipMemcpyDeviceToHost));
	return sum;
}
