#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include "cudaCommon.hu"
#include "cudaFolds.hu"

__device__ void devVecAdd(size_t pointDim, double* dest, double* src) {
	for(size_t i = 0; i < pointDim; ++i) {
		dest[i] += src[i];
	}
}

__global__ void kernElementWiseSum(const size_t numPoints, const size_t pointDim, double* dest, double* src) {
	// Assumes a 2D grid of 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;

	if(i < numPoints) {
		devVecAdd(pointDim, &dest[i * pointDim], &src[i * pointDim]);
	}
}

__global__ void kernBlockWiseSum(const size_t numPoints, const size_t pointDim, double* dest) {
	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;
	/*
	for(int s = blockDim.x / 2; threadIdx.x < s; s /= 2) {
	       devVecAdd(pointDim, &dest[i * pointDim], &dest[(i + s) * pointDim]);
		__syncthreads();
	} */

	// call repeatedly for each dimension where dest is assumed to begin at dimension d

	__shared__ double blockSum[1024];

	if(threadIdx.x >= numPoints) {
		blockSum[threadIdx.x] = 0;
	} else {
		blockSum[threadIdx.x] = dest[i * pointDim];
	}

	__syncthreads();

	// Do all the calculations in block shared memory instead of global memory.
	for(int s = blockDim.x / 2; threadIdx.x < s; s /= 2) {
		blockSum[threadIdx.x] += blockSum[threadIdx.x + s];
		__syncthreads();
	}

	if(threadIdx.x == 0) {
		// Just do one global write
		dest[i * pointDim] = blockSum[0];
	}	
}

__global__ void kernMoveMem(const size_t numPoints, const size_t pointDim, const size_t s, double* A) {
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;

	// Before
	// [abc......] [def......] [ghi......] [jkl......]

	// shared memory
	// [adgj.....]

	// After
	// [a..d..g..] [j........] [ghi......] [.........]

	__shared__ double mem[1024];
	mem[threadIdx.x] = A[s * i * pointDim];
	__syncthreads();
	A[i * pointDim] = mem[threadIdx.x];
}

__host__ void cudaArraySum(hipDeviceProp_t* deviceProp, size_t numPoints, const size_t pointDim, double* device_A, hipStream_t stream) {
	assert(deviceProp != NULL);
	assert(numPoints > 0);
	assert(pointDim > 0);
	assert(device_A != NULL);

	size_t M = largestPowTwoLessThanEq(numPoints);
	if(M != numPoints) {
		dim3 block , grid;
		calcDim(M, deviceProp, &block, &grid);
		kernElementWiseSum<<<grid, block, 0, stream>>>(
			numPoints - M, pointDim, device_A, device_A + M * pointDim
		);
		numPoints = M;
	}

	while(numPoints > 1) {
		dim3 block, grid;
		calcDim(numPoints, deviceProp, &block, &grid);

		for(size_t d = 0; d < pointDim; ++d) {
			kernBlockWiseSum<<<grid, block, 0, stream>>>(numPoints, pointDim, device_A + d);
			
			if(numPoints > block.x) {
				dim3 block2, grid2;
				calcDim(grid.x, deviceProp, &block2, &grid2);
				kernMoveMem<<<grid2, block2, 0, stream>>>(numPoints, pointDim, block.x, device_A + d);
			}
		}

		numPoints /= block.x;
	}

	/*
	// Parallel sum by continually folding the array in half and adding the right 
	// half to the left half until the fold size is 1024 (single block), then let
	// GPU reduce the remaining block to a single value and copy it over. O(log n).
	if(numPoints > 1024) {
		dim3 block, grid;
		for(numPoints /= 2; numPoints >= 1024; numPoints /= 2) {
			calcDim(numPoints, deviceProp, &block, &grid);
			kernElementWiseSum<<<grid, block, 0, stream>>>(
				numPoints, pointDim, device_A, device_A + numPoints * pointDim
			);
		}
		numPoints *= 2;
	}

	assert(numPoints <= 1024);

	for(size_t d = 0; d < pointDim; ++d) {
		kernBlockWiseSum<<<1, numPoints, 0, stream>>>(
			numPoints, pointDim, device_A + d
		);
	}
	*/
}

__host__ void cudaArraySum(hipDeviceProp_t* deviceProp, size_t numPoints, const size_t pointDim, double* device_A, double* host_sum) {
	assert(host_sum != NULL);
	check((void)0);
	cudaArraySum(deviceProp, numPoints, pointDim, device_A);
	check((void)0);
	check(hipMemcpy(host_sum, device_A, pointDim * sizeof(double), hipMemcpyDeviceToHost));
}

__global__ void kernElementWiseMax(int N, double* dest, double* src) {
	// Assumes a 2D grid of 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;
	if(i < N) {
		if(dest[i] < src[i]) {
			dest[i] = src[i];
		}
	}
}

__global__ void kernBlockWiseMax(const size_t N, double* dest) {
	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;

	// Load into block shared memory
	__shared__ double blockMax[1024];

	if(threadIdx.x >= N) {
		blockMax[threadIdx.x] = -INFINITY;
	} else {
		blockMax[threadIdx.x] = dest[i];
	}

	__syncthreads();	

	// Do all the calculations in block shared memory instead of global memory.
	for(int s = blockDim.x / 2; threadIdx.x < s; s /= 2) {
		if(blockMax[threadIdx.x] < blockMax[threadIdx.x + s]) {
			blockMax[threadIdx.x] = blockMax[threadIdx.x + s];
		}
		__syncthreads();
	}

	if(threadIdx.x == 0) {
		// Just do one global write instead of 2048.
		dest[i] = blockMax[0];
	}
}

__host__ double cudaArrayMax(hipDeviceProp_t* deviceProp, const size_t N, double* device_A) {
	// Parallel max by continually folding the array in half and maxing the right 
	// half to the left half until the fold size is 1024 (single block), then let
	// GPU reduce the remaining block to a single value and copy it over. O(log n).
	if(N > 1024) {
		assertPowerOfTwo(N);
		dim3 block, grid;
		for(size_t n = N/2; n >= 1024; n /= 2) {
			calcDim(n, deviceProp, &block, &grid);
			kernElementWiseMax<<<grid, block>>>(n, device_A, device_A + n);
			check((void)0);
		}
	}
 
	kernBlockWiseMax<<<1, 1024>>>(N, device_A);
	check((void)0);

	double maxValue = 0;
	check(hipMemcpy(&maxValue, device_A, sizeof(double), hipMemcpyDeviceToHost));
	check(hipDeviceSynchronize());
	return maxValue;
}
