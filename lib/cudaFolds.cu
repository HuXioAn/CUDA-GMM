#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include "cudaCommon.hu"
#include "cudaFolds.hu"

__device__ void devVecAdd(size_t pointDim, double* dest, double* src) {
	for(size_t i = 0; i < pointDim; ++i) {
		dest[i] += src[i];
	}
}

__global__ void kernElementWiseSum(const size_t numPoints, const size_t pointDim, double* dest, double* src) {
	// Assumes a 2D grid of 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;

	if(i < numPoints) {
		devVecAdd(pointDim, &dest[i * pointDim], &src[i * pointDim]);
	}
}

__global__ void kernBlockWiseSum(const size_t numPoints, const size_t pointDim, double* dest) {
	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;

	if(i < numPoints) {
		for(int s = blockDim.x / 2; threadIdx.x < s; s /= 2) {
			devVecAdd(pointDim, &dest[i * pointDim], &dest[(i + s) * pointDim]);
			__syncthreads();
		}
	}
}


__host__ void cudaArraySum(hipDeviceProp_t* deviceProp, size_t numPoints, const size_t pointDim, double* device_A, hipStream_t stream) {
	assert(deviceProp != NULL);
	assert(numPoints > 0);
	assert(pointDim > 0);
	assert(device_A != NULL);

	size_t M = largestPowTwoLessThanEq(numPoints);
	if(M != numPoints) {
		dim3 block , grid;
		calcDim(M, deviceProp, &block, &grid);
		kernElementWiseSum<<<grid, block, 0, stream>>>(
			numPoints - M, pointDim, device_A, device_A + M * pointDim
		);
		numPoints = M;
	}

	// Parallel sum by continually folding the array in half and adding the right 
	// half to the left half until the fold size is 1024 (single block), then let
	// GPU reduce the remaining block to a single value and copy it over. O(log n).
	if(numPoints > 1024) {
		dim3 block, grid;
		for(numPoints /= 2; numPoints >= 1024; numPoints /= 2) {
			calcDim(numPoints, deviceProp, &block, &grid);
			kernElementWiseSum<<<grid, block, 0, stream>>>(
				numPoints, pointDim, device_A, device_A + numPoints * pointDim
			);
		}
		numPoints *= 2;
	}

	assert(numPoints <= 1024);

	kernBlockWiseSum<<<1, numPoints, 0, stream>>>(
		numPoints, pointDim, device_A
	);
}

__host__ void cudaArraySum(hipDeviceProp_t* deviceProp, size_t numPoints, const size_t pointDim, double* device_A, double* host_sum) {
	assert(host_sum != NULL);
	cudaArraySum(deviceProp, numPoints, pointDim, device_A);
	check(hipMemcpy(host_sum, device_A, pointDim * sizeof(double), hipMemcpyDeviceToHost));
}

__global__ void kernElementWiseMax(int N, double* dest, double* src) {
	// Assumes a 2D grid of 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;
	if(i < N) {
		if(dest[i] < src[i]) {
			dest[i] = src[i];
		}
	}
}

__global__ void kernBlockWiseMax(const size_t N, double* dest) {
	// Assumes a 2D grid of 1024x1 1D blocks
	int b = blockIdx.y * gridDim.x + blockIdx.x;
	int i = b * blockDim.x + threadIdx.x;

	// Load into block shared memory
	__shared__ double blockMax[1024];

	if(threadIdx.x >= N) {
		blockMax[threadIdx.x] = -INFINITY;
	} else {
		blockMax[threadIdx.x] = dest[i];
	}

	__syncthreads();	

	// Do all the calculations in block shared memory instead of global memory.
	for(int s = blockDim.x / 2; threadIdx.x < s; s /= 2) {
		if(blockMax[threadIdx.x] < blockMax[threadIdx.x + s]) {
			blockMax[threadIdx.x] = blockMax[threadIdx.x + s];
		}
		__syncthreads();
	}

	if(threadIdx.x == 0) {
		// Just do one global write instead of 2048.
		dest[i] = blockMax[0];
	}
}

__host__ double cudaArrayMax(hipDeviceProp_t* deviceProp, const size_t N, double* device_A) {
	// Parallel max by continually folding the array in half and maxing the right 
	// half to the left half until the fold size is 1024 (single block), then let
	// GPU reduce the remaining block to a single value and copy it over. O(log n).
	if(N > 1024) {
		assertPowerOfTwo(N);
		dim3 block, grid;
		for(size_t n = N/2; n >= 1024; n /= 2) {
			calcDim(n, deviceProp, &block, &grid);
			kernElementWiseMax<<<grid, block>>>(n, device_A, device_A + n);
			check((void)0);
		}
	}
 
	kernBlockWiseMax<<<1, 1024>>>(N, device_A);
	check((void)0);

	double maxValue = 0;
	check(hipMemcpy(&maxValue, device_A, sizeof(double), hipMemcpyDeviceToHost));
	check(hipDeviceSynchronize());
	return maxValue;
}
