#include "hip/hip_runtime.h"
#include <assert.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>

// Intentionally not including header since it is meant for gcc consumption.
// #include "cudaWrappers.h"

#include "cudaCommon.hu"
#include "cudaFolds.hu"
#include "cudaGmm.hu"
#include "cudaMVNormal.hu"

extern "C" void gpuSum(size_t numPoints, const size_t pointDim, double* host_a, double* host_sum) {
	assert(numPoints > 0);
	assert(pointDim > 0);
	assert(host_a != NULL);
	assert(host_sum != NULL);

	int deviceId;
	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	// cudaArraySum is meant for powers of two 
	size_t M = largestPowTwoLessThanEq(numPoints);

	double cpuSum[pointDim];
	memset(cpuSum, 0, pointDim * sizeof(double));
	for(size_t i = M; i < numPoints; ++i) {
		for(size_t j = 0; j < pointDim; ++j) {
			cpuSum[j] += host_a[i * pointDim + j];
		}
	}

	numPoints = M;

	double *device_a = sendToGpu(numPoints * pointDim, host_a);

	// cudaArraySum is synchronous
	cudaArraySum(
		&deviceProp, numPoints, pointDim, device_a
		);

	check(hipMemcpy(host_sum, device_a, pointDim * sizeof(double), hipMemcpyDeviceToHost));

	hipFree(device_a);

	for(size_t i = 0; i < pointDim; ++i) {
		host_sum[i] += cpuSum[i];
	}
}

extern "C" double gpuMax(size_t N, double* host_a) {
	assert(host_a != NULL);
	assert(N > 0);

	int deviceId;
	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	double *device_a = sendToGpu(N, host_a);

	cudaArrayMax(
		&deviceProp, N, device_a
		);

	double gpuMax = 0;
	check(hipMemcpy(&gpuMax, device_a, sizeof(double), hipMemcpyDeviceToHost));

	hipFree(device_a);

	return gpuMax;
}

extern "C" void gpuLogMVNormDist(
	const size_t numPoints, const size_t pointDim,
	const double* X, const double* mu, const double* sigmaL,
	double* logP
) {
	int deviceId;
	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	double* device_X = sendToGpu(numPoints * pointDim, X);
	double* device_mu = sendToGpu(pointDim, mu);
	double* device_sigmaL = sendToGpu(pointDim * pointDim, sigmaL);
	double* device_logP = mallocOnGpu(numPoints);

	dim3 grid, block;
	calcDim(numPoints, &deviceProp, &block, &grid);
	kernLogMVNormDist<<<grid, block>>>(
		numPoints, pointDim,
		device_X, device_mu, device_sigmaL,
		device_logP
		);

	check(hipMemcpy(logP, device_logP, numPoints * sizeof(double), hipMemcpyDeviceToHost));

	hipDeviceSynchronize();

	hipFree(device_X);
	hipFree(device_mu);
	hipFree(device_sigmaL);
	hipFree(device_logP);
}

extern "C" double gpuGmmLogLikelihood(
	const size_t numPoints, const size_t numComponents,
	const double* logpi, double* logP
) {
	int deviceId;
	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	double* device_logpi = sendToGpu(numComponents, logpi);
	
	// Sending all data because logP is an array organized by:
	// [ <- numPoints -> ]_0 [ <- numPoints -> ]_... [ <- numPoints -> ]_{k-1}
	// So even though we are only using M of those points on the GPU,
	// we need all numPoints to ensure indexing by numPoints * k + i works
	// correctly to access prob(x_i|mu_k,Sigma_k).
	double* device_logP = sendToGpu(numComponents * numPoints, logP);

	double logL = cudaGmmLogLikelihoodAndGammaNK(
		& deviceProp,
		numPoints, numComponents,
		logpi, logP,
		device_logpi, device_logP
	);

	hipFree(device_logpi);
	hipFree(device_logP);

	return logL;
}

extern "C" void gpuCalcLogGammaNK(
	const size_t numPoints, const size_t numComponents,
	const double* logpi, double* loggamma
) { 
	gpuGmmLogLikelihood(
		numPoints, numComponents,
		logpi, loggamma
	); 
}

extern "C" void gpuCalcLogGammaK(
	const size_t numPoints, const size_t numComponents,
	const double* loggamma, double* logGamma
) {
	// Gamma[k] = max + log sum exp(loggamma - max)

	double* working = (double*)malloc(numPoints * sizeof(double));
	for(size_t k = 0; k < numComponents; ++k) {
		// TODO: refactor to have a generic z = a + log sum exp(x - a)
		memcpy(working, & loggamma[k * numPoints], numPoints * sizeof(double));
		double maxValue = gpuMax(numPoints, working);

		memcpy(working, & loggamma[k * numPoints], numPoints * sizeof(double));
		for(size_t i = 0; i < numPoints; ++i) {
			working[i] = exp(working[i] - maxValue);
		}

		double sum = 0;
		gpuSum(numPoints, 1, working, & sum);
 		logGamma[k] = maxValue + log(sum );
	}
	free(working);
}


extern "C" void gpuGmmFit(
	const double* X,
	const size_t numPoints, 
	const size_t pointDim, 
	const size_t numComponents,
	double* pi,
	double* Mu,
	double* Sigma,
	double* SigmaL,
	double* normalizers,
	const size_t maxIterations
) {
	assert(X != NULL);
	assert(numPoints > 0);
	assert(pointDim > 0 && pointDim <= 1024);
	assert(numComponents > 0 && numComponents <= 1024);

	assert(pi != NULL);
	assert(Mu != NULL);
	assert(Sigma != NULL);
	assert(SigmaL != NULL);
	assert(normalizers != NULL);

	assert(maxIterations >= 1);

	int deviceId;

	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	// printf("name: %s\n", deviceProp.name);
	// printf("multiProcessorCount: %d\n", deviceProp.multiProcessorCount);
	// printf("concurrentKernels: %d\n", deviceProp.concurrentKernels);

	double* device_X = pinHostAndSendDevice(numPoints * pointDim, (double*) X);

	for(size_t i = 0; i < numComponents; ++i) {
		assert(pi[i] > 0);
		pi[i] = log(pi[i]);
	}

	double* device_logpi = pinHostAndSendDevice(numComponents, pi);
	double* device_Mu = pinHostAndSendDevice(numComponents * pointDim, Mu);
	double* device_Sigma = pinHostAndSendDevice(numComponents * pointDim * pointDim, Sigma);

	double* device_SigmaL = pinHostAndSendDevice(numComponents * pointDim * pointDim, SigmaL);
	double* device_normalizers = pinHostAndSendDevice(numComponents, normalizers);

	double* device_loggamma = mallocOnGpu(numPoints * numComponents);
	double* device_logGamma = mallocOnGpu(numPoints * numComponents);	

	double previousLogL = -INFINITY;

	double* pinnedCurrentLogL;
	hipHostMalloc(&pinnedCurrentLogL, sizeof(double));
	*pinnedCurrentLogL = -INFINITY;

	// logPx, mu, sigma reductions
	// This means for mu and sigma can only do one component at a time otherwise 
	// the memory foot print will limit how much data we can actually work with.
	double* device_working = mallocOnGpu(numComponents * numPoints * pointDim * pointDim);

	dim3 grid, block;
	calcDim(numPoints, &deviceProp, &block, &grid);

	size_t iteration = 0;
	const double tolerance = 1e-8;

	hipStream_t streams[numComponents];
	for(size_t k = 0; k < numComponents; ++k) {
		hipStreamCreate(&streams[k]);
	}

	hipEvent_t kernelEvent[numComponents];
	for(size_t k = 0; k < numComponents; ++k) {
		hipEventCreateWithFlags(& kernelEvent[k], hipEventDisableTiming);
	}

	do {
		// --------------------------------------------------------------------------
		// E-Step
		// --------------------------------------------------------------------------

		// loggamma[k * numPoints + i] = p(x_i | mu_k, Sigma_k )
		for(size_t k = 0; k < numComponents; ++k) {
			// Fill in numPoint many probabilities
			kernLogMVNormDist<<<grid, block, 0, streams[k]>>>(
				numPoints, pointDim,
				device_X, 
				& device_Mu[k * pointDim], 
				& device_SigmaL[k * pointDim * pointDim],
				& device_loggamma[k * numPoints]
			);

			hipEventRecord(kernelEvent[k], streams[k]);
			hipStreamWaitEvent(streams[numComponents-1], kernelEvent[k], 0);
		}


		// loggamma[k * numPoints + i] = p(x_i | mu_k, Sigma_k) / p(x_i)
		// working[i] = p(x_i)
		kernCalcLogLikelihoodAndGammaNK<<<grid, block, 0, streams[numComponents - 1]>>>(
			numPoints, numComponents,
			device_logpi, device_working, device_loggamma
		);

		// working[0] = sum_{i} p(x_i)
		cudaArraySum(&deviceProp, numPoints, 1, device_working, streams[numComponents - 1]);

		previousLogL = *pinnedCurrentLogL;
		check(hipMemcpyAsync(
			pinnedCurrentLogL, device_working, 
			sizeof(double), 
			hipMemcpyDeviceToHost,
			streams[numComponents - 1]
		));

		hipEventRecord(kernelEvent[numComponents - 1], streams[numComponents - 1]);
		hipEventSynchronize(kernelEvent[numComponents - 1]);

		for(size_t k = 0; k < numComponents; ++k) {
			hipStreamSynchronize(streams[k]);
		}
		
		if(fabs(*pinnedCurrentLogL - previousLogL) < tolerance || *pinnedCurrentLogL < previousLogL) {
			break;
		}

		// --------------------------------------------------------------------------
		// M-Step
		// --------------------------------------------------------------------------

		for(size_t k = 0; k < numComponents; ++k) {
			cudaLogSumExp(
				& deviceProp, grid, block, 
				numPoints,
				& device_loggamma[k * numPoints], & device_logGamma[k * numPoints], 
				& device_working[k * numPoints], 
				streams[k]
			);
		}

		for(size_t k = 0; k < numComponents; ++k) {
			// working[i * pointDim + j] = gamma_ik / Gamma K * x_j
			kernCalcMu<<<grid, block, 0, streams[k]>>>(
				numPoints, pointDim,
				device_X, 
				& device_loggamma[k * numPoints], 
				& device_logGamma[k * numPoints],
				& device_working[k * numPoints * pointDim]
			);
		}

		for(size_t k = 0; k < numComponents; ++k) {
			// working[0 + j] = sum gamma_ik / Gamma K * x_j
			cudaArraySum(&deviceProp, numPoints, pointDim, & device_working[k * numPoints * pointDim], streams[k]);
		}

		for(size_t k = 0; k < numComponents; ++k) {
			check(hipMemcpyAsync(
				& device_Mu[k * pointDim],
				& device_working[k * pointDim * numPoints],
				pointDim * sizeof(double),
				hipMemcpyDeviceToDevice,
				streams[k]
			));
		}

		for(size_t k = 0; k < numComponents; ++k) {
			check(hipMemcpyAsync(
				& device_Sigma[k * pointDim * pointDim],
				& device_working[k * pointDim * pointDim * numPoints],
				pointDim * pointDim * sizeof(double),
				hipMemcpyDeviceToDevice,
				streams[k]
			));
		}

		for(size_t k = 0; k < numComponents; ++k) {
			// working[i * pointDim * pointDim + j] = 
			// 	gamma_ik / Gamma_k [ (x_i - mu) (x_i - mu)^T ]_j
			kernCalcSigma<<<grid, block, 0, streams[k]>>>(
				numPoints, pointDim,
				device_X, 
				& device_Mu[k * pointDim],
				& device_loggamma[k * numPoints],
				& device_logGamma[k * numPoints],
				& device_working[k * pointDim * pointDim * numPoints]
			);
		}

		for(size_t k = 0; k < numComponents; ++k) {
			// working[0 + j] = sum gamma_ik / Gamma K * [...]_j
			cudaArraySum(
				&deviceProp, numPoints, pointDim * pointDim, 
				&device_working[k * pointDim * pointDim * numPoints], streams[k]
			);
		}

		for(size_t k = 0; k < numComponents; ++k) {
			check(hipMemcpyAsync(
				& device_Sigma[k * pointDim * pointDim],
				& device_working[k * pointDim * pointDim * numPoints],
				pointDim * pointDim * sizeof(double),
				hipMemcpyDeviceToDevice,
				streams[k]
			));

			hipEventRecord(kernelEvent[k], streams[k]);
			hipStreamWaitEvent(streams[numComponents-1], kernelEvent[k], 0);
		}

		// pi_k^(t+1) = pi_k Gamma_k / sum_{i}^{K} pi_i * Gamma_i
		// Use thread sync to compute denom to avoid data race
		kernUpdatePi<<<1, numComponents, 0, streams[numComponents - 1]>>>(
			numPoints, numComponents,
			device_logpi, device_logGamma
		);

		// recompute sigmaL and normalizer
		kernPrepareCovariances<<<1, numComponents, 0, streams[numComponents - 1]>>>(
			numComponents, pointDim,
			device_Sigma, device_SigmaL,
			device_normalizers
		);

		hipEventRecord(kernelEvent[numComponents - 1], streams[numComponents - 1]);
		hipStreamWaitEvent(streams[numComponents - 1], kernelEvent[numComponents - 1], 0);

		for(size_t k = 0; k < numComponents; ++k) {
			hipStreamSynchronize(streams[k]);
		}
	} while(++iteration < maxIterations);

	for(size_t k = 0; k < numComponents; ++k) {
		hipEventDestroy(kernelEvent[k]);
	}

	for(size_t k = 0; k < numComponents; ++k) {
		hipStreamDestroy(streams[k]);
	}

	hipHostFree(pinnedCurrentLogL);
	hipFree(device_working);
	hipFree(device_logGamma);
	hipFree(device_loggamma);

	recvDeviceUnpinHost(device_normalizers, normalizers, numComponents);
	recvDeviceUnpinHost(device_SigmaL, SigmaL, numComponents * pointDim * pointDim);
	recvDeviceUnpinHost(device_Sigma, Sigma, numComponents * pointDim * pointDim);
	recvDeviceUnpinHost(device_Mu, Mu, numComponents * pointDim);
	recvDeviceUnpinHost(device_logpi, pi, numComponents);

	for(size_t i = 0; i < numComponents; ++i) {
		pi[i] = exp(pi[i]);
	}

	unpinHost(device_X, (double*) X);
}
