#include "hip/hip_runtime.h"
#include <assert.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>

// Intentionally not including header since it is meant for gcc consumption.
// #include "cudaWrappers.h"
#include "cudaKernels.hu"

#define check(call) { hipError_t __ABC123 = call; assert(__ABC123 == hipSuccess); }

extern "C" void parallelGmmLogLikelihood(
	const size_t numPoints, const size_t numComponents,
	const double* logPi, double* logProb
) {
	int deviceId;
	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	double* device_logPi;
	const size_t logPiBytes = numComponents * sizeof(double);
	check(hipMalloc(&device_logPi, logPiBytes));
	check(hipMemcpy(device_logPi, logPi, logPiBytes, hipMemcpyHostToDevice));

	double* device_logProb;
	const size_t logProbBytes = numComponents * numPoints * sizeof(double);
	check(hipMalloc(&device_logProb, logProbBytes));
	check(hipMemcpy(device_logProb, logProb, logProbBytes, hipMemcpyHostToDevice));

	cudaGmmLogLikelihood<<<numPoints, 1>>>(
		numPoints, numComponents, 
		device_logPi, device_logProb
	);

	// Not final action; need to do parallelSum
	check(hipMemcpy(logProb, device_logProb, numPoints * sizeof(double), hipMemcpyDeviceToHost));

	hipDeviceSynchronize();
	
	hipFree(device_logPi);
	hipFree(device_logProb);
}

extern "C" void parallelLogMVNormDist(
	const size_t numPoints, const size_t pointDim,
	const double* X, const double* mu, const double* sigmaL, const double logNormalizer,
	double* logP
) {
	int deviceId;
	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	double* device_sigmaL;
	const size_t sigmaLBytes = pointDim * pointDim * sizeof(double);
	check(hipMalloc(&device_sigmaL, sigmaLBytes));
	check(hipMemcpy(device_sigmaL, sigmaL, sigmaLBytes, hipMemcpyHostToDevice));

	double* device_mu;
	const size_t muBytes = pointDim * sizeof(double);
	check(hipMalloc(&device_mu, muBytes));
	check(hipMemcpy(device_mu, mu, muBytes, hipMemcpyHostToDevice));

	double* device_X;
	const size_t XBytes = numPoints * pointDim * sizeof(double);
	check(hipMalloc(&device_X, XBytes));
	check(hipMemcpy(device_X, X, XBytes, hipMemcpyHostToDevice));

	double* device_logP;
	double logPBytes = numPoints * sizeof(double);
	check(hipMalloc(&device_logP, logPBytes));

	cudaLogMVNormDist<<<numPoints, 1>>>(
		numPoints, pointDim,
		device_X, device_mu, device_sigmaL, logNormalizer,
		device_logP
		);

	check(hipMemcpy(logP, device_logP, logPBytes, hipMemcpyDeviceToHost));

	hipDeviceSynchronize();

	hipFree(device_sigmaL);
	hipFree(device_mu);
	hipFree(device_X);
	hipFree(device_logP);
}

extern "C" double parallelSum(double* host_a, const size_t N) {
	assert(host_a != NULL);
	assert(N > 0);
	assertPowerOfTwo(N);

	const size_t NBytes = N * sizeof(double);

	int deviceId;
	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	double *device_a;
	check(hipMalloc(&device_a, NBytes));
	check(hipMemcpy(device_a, host_a, NBytes, hipMemcpyHostToDevice));

	// Parallel sum by continually folding the array in half and adding the right 
	// half to the left half until the fold size is 1024 (single block), then let
	// GPU reduce the remaining block to a single value and copy it over. O(log n).
	if(N >= 1024) {
		dim3 block, grid;
		for(size_t n = N/2; n >= 1024; n /= 2) {
			calcDim(n, &deviceProp, &block, &grid);
			cudaSum<<<grid, block>>>(n, device_a, device_a + n);
		}
		cudaReduceBlocks<<<1, 1024>>>(device_a);
	} else {
		cudaReduceBlocks<<<1, N>>>(device_a);
	}

	double host_sum = 0;
	check(hipMemcpy(&host_sum, device_a, sizeof(double), hipMemcpyDeviceToHost));

	hipDeviceSynchronize();

	hipFree(device_a);

	return host_sum;
}

