#include "hip/hip_runtime.h"
#include <assert.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>

// Intentionally not including header since it is meant for gcc consumption.
// #include "cudaWrappers.h"

#include "cudaCommon.hu"
#include "cudaMVNormal.hu"
#include "cudaGmm.hu"

extern "C" void gpuSum(size_t numPoints, const size_t pointDim, double* host_a, double* host_sum) {
	assert(numPoints > 0);
	assert(pointDim > 0);
	assert(host_a != NULL);
	assert(host_sum != NULL);

	int deviceId;
	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	// cudaArraySum is meant for powers of two 
	double cpuSum[pointDim];
	memset(cpuSum, 0, pointDim * sizeof(double));

	size_t M = 2;
	while(M < numPoints) {
		M *= 2;
	}

	if(M > numPoints) {
		M /= 2;
		for(size_t i = M; i < numPoints; ++i) {
			for(size_t j = 0; j < pointDim; ++j) {
				cpuSum[j] += host_a[i * pointDim + j];
			}
		}
		numPoints = M;
	}

	double *device_a = sendToGpu(numPoints * pointDim, host_a);

	// cudaArraySum is synchronous
	cudaArraySum(
		&deviceProp, numPoints, pointDim, device_a, host_sum
		);

	hipFree(device_a);

	for(size_t i = 0; i < pointDim; ++i) {
		host_sum[i] += cpuSum[i];
	}
}

extern "C" double gpuMax(size_t N, double* host_a) {
	assert(host_a != NULL);
	assert(N > 0);

	int deviceId;
	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	double cpuMax = -INFINITY;
	if(N > 1024) {
		// cudaArrayMax is meant for powers of two when N > 1024; 
		size_t M = 2;
		while(M < N) {
			M *= 2;
		}

		if(M > N) {
			M /= 2;
			for(size_t i = M; i < N; ++i) {
				if(host_a[i] > cpuMax) {
					cpuMax = host_a[i];
				}
			}
			N = M;
		}
	}

	double *device_a = sendToGpu(N, host_a);

	// cudaArrayMax is synchronous
	double gpuMax = cudaArrayMax(
		&deviceProp, N, device_a
		);

	hipFree(device_a);

	if(cpuMax > gpuMax) {
		return cpuMax;
	}

	return gpuMax;
}

extern "C" void gpuLogMVNormDist(
	const size_t numPoints, const size_t pointDim,
	const double* X, const double* mu, const double* sigmaL,
	double* logP
) {
	int deviceId;
	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	double* device_X = sendToGpu(numPoints * pointDim, X);
	double* device_mu = sendToGpu(pointDim, mu);
	double* device_sigmaL = sendToGpu(pointDim * pointDim, sigmaL);
	double* device_logP = mallocOnGpu(numPoints);

	// TODO: calcDim...
	kernLogMVNormDist<<<1, numPoints>>>(
		numPoints, pointDim,
		device_X, device_mu, device_sigmaL,
		device_logP
		);

	check(hipMemcpy(logP, device_logP, numPoints * sizeof(double), hipMemcpyDeviceToHost));

	hipDeviceSynchronize();

	hipFree(device_X);
	hipFree(device_mu);
	hipFree(device_sigmaL);
	hipFree(device_logP);
}

extern "C" double gpuGmmLogLikelihood(
	const size_t numPoints, const size_t numComponents,
	const double* logPi, const double* logP
) {
	int deviceId;
	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	double* device_logPi = sendToGpu(numComponents, logPi);
	double* device_logP = sendToGpu(numComponents * numPoints, logP);
	double* device_logL = mallocOnGpu(numPoints);

	// TODO: calcDim...
	kernGmmLogLikelihood<<<1, numPoints>>>(
		numPoints, numComponents,
		device_logPi, device_logP, device_logL
	);

	// cudaArraySum is synchronous
	double logL = 0;
	cudaArraySum(&deviceProp, numPoints, 1, device_logL, &logL);

	hipFree(device_logPi);
	hipFree(device_logP);
	hipFree(device_logL);

	return logL;
}

extern "C" void gpuCalcLogGammaNK(
	const size_t numPoints, const size_t pointDim, const size_t numComponents,
	const double* logpi, double* loggamma
) { 
	int deviceId;
	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	double* device_logpi = sendToGpu(numComponents, logpi);
	double* device_loggamma = sendToGpu(numComponents * numPoints, loggamma);

	// TODO: calcDim...
	kernCalcLogGammaNK<<<1, numPoints>>>(
		numPoints, pointDim, numComponents,
		device_logpi, device_loggamma
	);

	hipMemcpy(loggamma, device_loggamma, numComponents * numPoints * sizeof(double), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	hipFree(device_logpi);
	hipFree(device_loggamma);
}

extern "C" void gpuCalcLogGammaK(
	const size_t numPoints, const size_t numComponents,
	const double* loggamma, double* logGamma
) {
	// Gamma[k] = max + log sum exp(loggamma - max)

	double* working = (double*)malloc(numPoints * sizeof(double));
	for(size_t k = 0; k < numComponents; ++k) {
		// TODO: refactor to have a generic z = a + log sum exp(x - a)
		memcpy(working, & loggamma[k * numPoints], numPoints * sizeof(double));
		double maxValue = gpuMax(numPoints, working);

		memcpy(working, & loggamma[k * numPoints], numPoints * sizeof(double));
		for(size_t i = 0; i < numPoints; ++i) {
			working[i] = exp(working[i] - maxValue);
		}

		double sum = 0;
		gpuSum(numPoints, 1, working, & sum);
 		logGamma[k] = maxValue + log(sum );
	}
	free(working);
}

extern "C" void gpuPerformMStep(
	const size_t numPoints, const size_t pointDim,
	const double* X, 
	double* loggamma, double logGammaK, double logGammaSum,
	double* logpik, double* mu, double* sigma
) {
	// X: pointDim x numPoints
	// loggamma: 1 x numPoints
	// logGamma: 1 x 1
	// logGammaSum: 1 x 1
	// logPi: 1 x 1
	// mu: 1 x pointDim
	// sigma: pointDim x pointDim

	*logpik += logGammaK - logGammaSum;

	double* device_X = sendToGpu(numPoints * pointDim, X);
	double* device_loggamma = sendToGpu(numPoints, loggamma);

	double* device_mu = mallocOnGpu(pointDim);
	double* device_mu_working = mallocOnGpu(numPoints * pointDim);

	// calcDim(...)

	// Calculate mu_+1
	kernCalcMu<<<1, numPoints>>>(
		numPoints, pointDim, 
		device_X, device_loggamma, logGammaSum, 
		device_mu_working
	);

	cudaVecArraySum(
		numPoints, pointDim, 
		device_mu_working
	);
		
	check(hipMemcpy(device_mu_working, device_mu, 
		pointDim* sizeof(double), hipMemcpyDeviceToDevice));

	hipFree(device_mu_working);

	hipDeviceSynchronize();

	// Calculate sigma_+1
	double* device_sigma_working = mallocOnGpu(numPoints * pointDim * pointDim);
	kernCalcSigma<<<1, numPoints>>>(
		numPoints, pointDim, 
		device_X, device_mu, device_loggamma, logGammaSum, 
		device_sigma_working
	);

	cudaVecArraySum(
		numPoints, pointDim, 
		device_sigma_working
	);

	check(hipMemcpy(sigma, device_sigma_working, 
		pointDim* pointDim*sizeof(double), hipMemcpyDeviceToHost));
	hipFree(device_sigma_working);

	check(hipMemcpy(mu, device_mu,
		pointDim * sizeof(double), hipMemcpyDeviceToHost));
	hipFree(device_mu);

	hipFree(device_X);
	hipFree(device_loggamma);

	// doing the cholesky decomposition is caller (cpu) side for now
}
