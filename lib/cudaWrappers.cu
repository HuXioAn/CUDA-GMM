#include <assert.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>

// Intentionally not including header since it is meant for gcc consumption.
// #include "cudaWrappers.h"

#include "cudaCommon.hu"
#include "cudaMVNormal.hu"
#include "cudaGmm.hu"

double* mallocOnGpu(const size_t N) {
	double* device_A;
	double ABytes = N * sizeof(double);
	check(hipMalloc(&device_A, ABytes));
	return device_A;
}

double* sendToGpu(const size_t N, const double* A) {
	double* device_A;
	const size_t ABytes = N * sizeof(double);
	check(hipMalloc(&device_A, ABytes));
	check(hipMemcpy(device_A, A, ABytes, hipMemcpyHostToDevice));
	return device_A;
}

extern "C" void gpuLogMVNormDist(
	const size_t numPoints, const size_t pointDim,
	const double* X, const double* mu, const double* sigmaL,
	double* logP
) {
	int deviceId;
	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	double* device_X = sendToGpu(numPoints * pointDim, X);
	double* device_mu = sendToGpu(pointDim, mu);
	double* device_sigmaL = sendToGpu(pointDim * pointDim, sigmaL);
	double* device_logP = mallocOnGpu(numPoints);

	kernLogMVNormDist<<<numPoints, 1>>>(
		numPoints, pointDim,
		device_X, device_mu, device_sigmaL,
		device_logP
		);

	check(hipMemcpy(logP, device_logP, numPoints * sizeof(double), hipMemcpyDeviceToHost));

	hipDeviceSynchronize();

	hipFree(device_X);
	hipFree(device_mu);
	hipFree(device_sigmaL);
	hipFree(device_logP);
}

extern "C" double gpuGmmLogLikelihood(
	const size_t numPoints, const size_t numComponents,
	const double* logPi, const double* logP
) {
	int deviceId;
	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	// TODO: Power of two padding?
	double* device_logPi = sendToGpu(numComponents, logPi);
	double* device_logP = sendToGpu(numComponents, logP);
	double* device_logL = mallocOnGpu(numPoints);

	// TODO: calcDim...
	kernGmmLogLikelihood<<<numPoints, 1>>>(
		numPoints, numComponents,
		device_logPi, device_logP, device_logL
	);

	double logL = cudaReduceSum(&deviceProp, numPoints, device_logL);

	hipFree(device_logPi);
	hipFree(device_logP);
	hipFree(device_logL);

	return logL;
}

extern "C" double gpuSum(const size_t N, double* host_a) {
	assert(host_a != NULL);
	assert(N > 0);
	assertPowerOfTwo(N);

	int deviceId;
	check(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	check(hipGetDeviceProperties(&deviceProp, deviceId));

	double *device_a = sendToGpu(N, host_a);

	double sum = cudaReduceSum(
		&deviceProp, N, device_a
		);

	hipFree(device_a);

	return sum;
}

